#include "hip/hip_runtime.h"
#include <tune_quda.h>
#include <transfer.h>
#include <color_spinor_field.h>
#include <gauge_field.h>

#include <jitify_helper.cuh>

// For naive Kahler-Dirac coarsening
#include <kernels/staggered_coarse_op_kernel.cuh>

// This define controls which kernels get compiled in `coarse_op.cuh`.
// This ensures only kernels relevant for coarsening the staggered
// operator get built, saving compile time.
#define STAGGEREDCOARSE
#include <coarse_op.cuh>

namespace quda {

  /**
     @brief dummyClover is a helper function to allow us to create an
     empty clover object - this allows us to use the the externally
     linked reduction kernels when we do have a clover field. Taken from
     coarsecoarse_op.cu.
   */
  inline std::unique_ptr<cudaCloverField> dummyClover()
  {
    CloverFieldParam cf_param;
    cf_param.nDim = 4;
    cf_param.pad = 0;
    cf_param.setPrecision(QUDA_SINGLE_PRECISION);

    for (int i = 0; i < cf_param.nDim; i++) cf_param.x[i] = 0;

    cf_param.direct = true;
    cf_param.inverse = true;
    cf_param.clover = nullptr;
    cf_param.norm = 0;
    cf_param.cloverInv = nullptr;
    cf_param.invNorm = 0;
    cf_param.create = QUDA_NULL_FIELD_CREATE;
    cf_param.siteSubset = QUDA_FULL_SITE_SUBSET;

    // create a dummy cudaCloverField if one is not defined
    cf_param.order = QUDA_INVALID_CLOVER_ORDER;
    return std::make_unique<cudaCloverField>(cf_param);
  }

  template <typename Float, int fineColor, int coarseSpin, int coarseColor, typename Arg>
  class CalculateStaggeredY : public TunableVectorYZ {

    Arg &arg;
    const GaugeField &meta;
    GaugeField &Y;
    GaugeField &X;

    long long flops() const { return arg.coarseVolumeCB*coarseSpin*coarseColor; }

    long long bytes() const
    {
      // 2 from forwards / backwards contributions, Y and X are sparse - only needs to write non-zero elements, 2nd term is mass term
      return meta.Bytes() + (2 * meta.Bytes() * Y.Precision()) / meta.Precision() + 2 * 2 * coarseSpin * coarseColor * arg.coarseVolumeCB * X.Precision();
    }

    unsigned int minThreads() const { return arg.fineVolumeCB; }
    bool tuneSharedBytes() const { return false; } // don't tune the grid dimension
    bool tuneGridDim() const { return false; } // don't tune the grid dimension
    bool tuneAuxDim() const { return false; }

  public:
    CalculateStaggeredY(Arg &arg, const GaugeField &meta, GaugeField &Y, GaugeField &X) :
      TunableVectorYZ(fineColor*fineColor, 2),
      arg(arg),
      meta(meta),
      Y(Y),
      X(X)
    {
      if (meta.Location() == QUDA_CUDA_FIELD_LOCATION) {
#ifdef JITIFY
        create_jitify_program("kernels/staggered_coarse_op_kernel.cuh");
#endif
      }
      strcpy(aux, compile_type_str(meta));
      strcpy(aux, meta.AuxString());
      strcat(aux,comm_dim_partitioned_string());
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) strcat(aux, getOmpThreadStr());
      strcat(aux,",computeStaggeredVUV");
      strcat(aux, (meta.Location()==QUDA_CUDA_FIELD_LOCATION && Y.MemType() == QUDA_MEMORY_MAPPED) ? ",GPU-mapped," :
             meta.Location()==QUDA_CUDA_FIELD_LOCATION ? ",GPU-device," : ",CPU,");
      strcat(aux,"coarse_vol=");
      strcat(aux,X.VolString());
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), QUDA_VERBOSE);

      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
        ComputeStaggeredVUVCPU<Float,fineColor,coarseSpin,coarseColor>(arg);
      } else {
#ifdef JITIFY
        using namespace jitify::reflection;
        jitify_error = program->kernel("quda::ComputeStaggeredVUVGPU")
          .instantiate(Type<Float>(),fineColor,coarseSpin,coarseColor,Type<Arg>())
          .configure(tp.grid,tp.block,tp.shared_bytes,stream).launch(arg);
#else // not jitify
        qudaLaunchKernel(ComputeStaggeredVUVGPU<Float,fineColor,coarseSpin,coarseColor,Arg>, tp, stream, arg);
#endif // JITIFY
      }
    }

    bool advanceTuneParam(TuneParam &param) const {
      // only do autotuning if we have device fields
      if (meta.Location() == QUDA_CUDA_FIELD_LOCATION && Y.MemType() == QUDA_MEMORY_DEVICE) return Tunable::advanceTuneParam(param);
      else return false;
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }
  };

  /**
     @brief Calculate the coarse-link field, including the coarse clover field.

     @param Y[out] Coarse (fat-)link field accessor
     @param X[out] Coarse clover field accessor
     @param G[in] Fine grid link / gauge field accessor
     @param Y_[out] Coarse link field
     @param X_[out] Coarse clover field
     @param X_[out] Coarse clover inverese field (used as temporary here)
     @param v[in] Packed null-space vectors
     @param G_[in] Fine gauge field
     @param mass[in] Kappa parameter
     @param matpc[in] The type of preconditioning of the source fine-grid operator
   */
  template<typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor,
	   typename coarseGauge, typename fineGauge>
  void calculateStaggeredY(coarseGauge &Y, coarseGauge &X, fineGauge &G, GaugeField &Y_, GaugeField &X_,
                           const GaugeField &G_, double mass, QudaDiracType dirac, QudaMatPCType matpc)
  {
    // sanity checks
    if (matpc == QUDA_MATPC_EVEN_EVEN_ASYMMETRIC || matpc == QUDA_MATPC_ODD_ODD_ASYMMETRIC)
      errorQuda("Unsupported coarsening of matpc = %d", matpc);

    // This is the last time we use fineSpin, since this file only coarsens
    // staggered-type ops, not wilson-type AND coarse-type.
    if (fineSpin != 1)
      errorQuda("Input Dirac operator %d should have nSpin=1, not nSpin=%d\n", dirac, fineSpin);
    if (fineColor != 3)
      errorQuda("Input Dirac operator %d should have nColor=3, not nColor=%d\n", dirac, fineColor);

    if (G.Ndim() != 4) errorQuda("Number of dimensions not supported");
    const int nDim = 4;

    int x_size[QUDA_MAX_DIM] = { };
    for (int i=0; i<4; i++) x_size[i] = G_.X()[i];
    x_size[4] = 1;

    int xc_size[QUDA_MAX_DIM] = { };
    for (int i=0; i<4; i++) xc_size[i] = X_.X()[i];
    xc_size[4] = 1;

    int geo_bs[QUDA_MAX_DIM] = { };
    for(int d = 0; d < nDim; d++) geo_bs[d] = x_size[d]/xc_size[d];
    int spin_bs = 0; // 0 -> spin-less types.

    // Calculate VUV in one pass (due to KD-transform) for each dimension,
    // accumulating directly into the coarse gauge field Y

    using Arg = CalculateStaggeredYArg<Float,coarseSpin,fineColor,coarseColor,coarseGauge,fineGauge>;
    Arg arg(Y, X, G, mass, x_size, xc_size, geo_bs, spin_bs);
    CalculateStaggeredY<Float, fineColor, coarseSpin, coarseColor, Arg> y(arg, G_, Y_, X_);

    QudaFieldLocation location = checkLocation(Y_, X_, G_);
    if (getVerbosity() >= QUDA_VERBOSE) printfQuda("Running link coarsening on the %s\n", location == QUDA_CUDA_FIELD_LOCATION ? "GPU" : "CPU");

    // We know exactly what the scale should be: the max of all of the (fat) links.
    double max_scale = G_.abs_max();
    if (getVerbosity() >= QUDA_VERBOSE) printfQuda("Global U_max = %e\n", max_scale);

    if (coarseGauge::fixedPoint()) {
      arg.Y.resetScale(max_scale);
      arg.X.resetScale(max_scale > 2.0*mass ? max_scale : 2.0*mass); // To be safe
      Y_.Scale(max_scale);
      X_.Scale(max_scale > 2.0*mass ? max_scale : 2.0*mass); // To be safe
    }

    // We can technically do a uni-directional build, but becauase
    // the coarse link builds are just permutations plus lots of zeros,
    // it's faster to skip the flip!

    if (getVerbosity() >= QUDA_VERBOSE) printfQuda("Computing VUV\n");
    y.apply(0);

    if (getVerbosity() >= QUDA_VERBOSE) {
      for (int d = 0; d < nDim; d++) printfQuda("Y2[%d] = %e\n", 4+d, Y_.norm2( 4+d ));
      for (int d = 0; d < nDim; d++) printfQuda("Y2[%d] = %e\n", d, Y_.norm2( d ));
    }

    if (getVerbosity() >= QUDA_VERBOSE) printfQuda("X2 = %e\n", X_.norm2(0));
  }

  template <typename Float, typename vFloat, int fineColor, int fineSpin, int coarseColor, int coarseSpin>
  void calculateStaggeredY(GaugeField &Y, GaugeField &X, const Transfer &T, const GaugeField &g,
                           double mass, QudaDiracType dirac, QudaMatPCType matpc)
  {
    QudaFieldLocation location = Y.Location();

    if (location == QUDA_CPU_FIELD_LOCATION) {

      constexpr QudaFieldOrder csOrder = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;
      constexpr QudaGaugeFieldOrder gOrder = QUDA_QDP_GAUGE_ORDER;

      if (T.Vectors(Y.Location()).FieldOrder() != csOrder)
        errorQuda("Unsupported field order %d\n", T.Vectors(Y.Location()).FieldOrder());
      if (g.FieldOrder() != gOrder) errorQuda("Unsupported field order %d\n", g.FieldOrder());

      using gFine = typename gauge::FieldOrder<Float,fineColor,1,gOrder>;
      using gCoarse = typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder,true,vFloat>;

      gFine gAccessor(const_cast<GaugeField&>(g));
      gCoarse yAccessor(const_cast<GaugeField&>(Y));
      gCoarse xAccessor(const_cast<GaugeField&>(X));

      calculateStaggeredY<Float,fineSpin,fineColor,coarseSpin,coarseColor>
        (yAccessor, xAccessor, gAccessor, Y, X, g, mass, dirac, matpc);
    } else {

      constexpr QudaFieldOrder csOrder = QUDA_FLOAT2_FIELD_ORDER;
      constexpr QudaGaugeFieldOrder gOrder = QUDA_FLOAT2_GAUGE_ORDER;

      if (T.Vectors(Y.Location()).FieldOrder() != csOrder)
        errorQuda("Unsupported field order %d\n", T.Vectors(Y.Location()).FieldOrder());
      if (g.FieldOrder() != gOrder) errorQuda("Unsupported field order %d\n", g.FieldOrder());

      using gFine = typename gauge::FieldOrder<Float,fineColor,1,gOrder,true,Float>;
      using gCoarse = typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder,true,vFloat>;

      gFine gAccessor(const_cast<GaugeField&>(g));
      gCoarse yAccessor(const_cast<GaugeField&>(Y));
      gCoarse xAccessor(const_cast<GaugeField&>(X));

      calculateStaggeredY<Float,fineSpin,fineColor,coarseSpin,coarseColor>
        (yAccessor, xAccessor, gAccessor, Y, X, g, mass, dirac, matpc);
    }

  }

  template <typename Float, typename vFloat, int fineColor, int fineSpin, int coarseColor, int coarseSpin, int uvSpin>
  void aggregateStaggeredY(GaugeField &Y, GaugeField &X, const Transfer &T, const GaugeField &g, const GaugeField &l,
                            const GaugeField &XinvKD, double mass, QudaDiracType dirac, QudaMatPCType matpc)
  {
    // Actually create the temporaries like UV, etc.
    auto location = Y.Location();

    //Create a field UV which holds U*V.  Has the same structure as V,
    ColorSpinorParam UVparam(T.Vectors(location));
    UVparam.create = QUDA_ZERO_FIELD_CREATE;
    UVparam.location = location;
    UVparam.nSpin = uvSpin;
    UVparam.setPrecision(T.Vectors(location).Precision());
    UVparam.mem_type = Y.MemType(); // allocate temporaries to match coarse-grid link field

    ColorSpinorField *uv = ColorSpinorField::Create(UVparam);

    ColorSpinorField *av = (dirac == QUDA_STAGGEREDKD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC) ? ColorSpinorField::Create(UVparam) : &const_cast<ColorSpinorField&>(T.Vectors(location));
    
    GaugeField *Yatomic = &Y;
    GaugeField *Xatomic = &X;

    if (Y.Precision() < QUDA_SINGLE_PRECISION) {
      // we need to coarsen into single precision fields (float or int), so we allocate temporaries for this purpose
      // else we can just coarsen directly into the original fields
      GaugeFieldParam param(X); // use X since we want scalar geometry
      param.location = location;
      param.setPrecision(QUDA_SINGLE_PRECISION, location == QUDA_CUDA_FIELD_LOCATION ? true : false);

      Yatomic = GaugeField::Create(param);
      Xatomic = GaugeField::Create(param);
    }

    // Moving along to the build

    const double kappa = -1.; // cancels a minus sign factor for kappa w/in the dslash application
    const double mu_dummy = 0.; 
    const double mu_factor_dummy = 0.;

    bool need_bidirectional = false;
    if (dirac == QUDA_STAGGEREDKD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC) need_bidirectional = true;

    const int nFace = (dirac == QUDA_ASQTAD_DIRAC || dirac == QUDA_ASQTADPC_DIRAC || dirac == QUDA_ASQTADKD_DIRAC) ? 3 : 1;

    if (Y.Location() == QUDA_CPU_FIELD_LOCATION) {

      constexpr QudaFieldOrder csOrder = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;
      constexpr QudaGaugeFieldOrder gOrder = QUDA_QDP_GAUGE_ORDER;

      if (T.Vectors(Y.Location()).FieldOrder() != csOrder)
        errorQuda("Unsupported field order %d\n", T.Vectors(Y.Location()).FieldOrder());
      if (g.FieldOrder() != gOrder) errorQuda("Unsupported field order %d\n", g.FieldOrder());

      using V = typename colorspinor::FieldOrderCB<Float,fineSpin,fineColor,coarseColor,csOrder,vFloat>;
      using F = typename colorspinor::FieldOrderCB<Float,uvSpin,fineColor,coarseColor,csOrder,vFloat>;
      using gFine = typename gauge::FieldOrder<Float,fineColor,1,gOrder>;
      using gCoarse = typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder,true,vFloat>;
      using gCoarseAtomic = typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder,true,storeType>;

      const ColorSpinorField &v = T.Vectors(Y.Location());

      V vAccessor(const_cast<ColorSpinorField&>(v), nFace);
      F uvAccessor(*uv, nFace); // will need 2x the spin components for the KD op
      F avAccessor(*av, nFace);
      gFine gAccessor(const_cast<GaugeField&>(g));
      gFine lAccessor(const_cast<GaugeField&>(g));
      gFine xinvAccessor(const_cast<GaugeField&>(XinvKD));
      gCoarse yAccessor(const_cast<GaugeField&>(Y));
      gCoarse xAccessor(const_cast<GaugeField&>(X));
      gCoarseAtomic yAccessorAtomic(*Yatomic);
      gCoarseAtomic xAccessorAtomic(*Xatomic);

      // the repeated xinvAccessor is intentional
      calculateY<QUDA_CPU_FIELD_LOCATION, false, Float, fineSpin, fineColor, coarseSpin, coarseColor>(
        yAccessor, xAccessor, yAccessorAtomic, xAccessorAtomic, uvAccessor, avAccessor, vAccessor, gAccessor, lAccessor, xinvAccessor, xAccessor, xAccessor,
        Y, X, *Yatomic, *Xatomic, *uv, *av, v, g, l, XinvKD, *dummyClover(), kappa, mass, mu_dummy,
        mu_factor_dummy, dirac, matpc, need_bidirectional, T.fineToCoarse(Y.Location()), T.coarseToFine(Y.Location()));
    } else {

      constexpr QudaFieldOrder csOrder = QUDA_FLOAT2_FIELD_ORDER;
      constexpr QudaGaugeFieldOrder gOrder = QUDA_FLOAT2_GAUGE_ORDER;

      if (T.Vectors(Y.Location()).FieldOrder() != csOrder)
        errorQuda("Unsupported field order %d\n", T.Vectors(Y.Location()).FieldOrder());
      if (g.FieldOrder() != gOrder) errorQuda("Unsupported field order %d\n", g.FieldOrder());

      using V = typename colorspinor::FieldOrderCB<Float, fineSpin, fineColor, coarseColor, csOrder, vFloat, vFloat, false, false>;
      using F = typename colorspinor::FieldOrderCB<Float, uvSpin, fineColor, coarseColor, csOrder, vFloat, vFloat, false, false>; // will need 2x the spin components for the KD op
      using gFine =  typename gauge::FieldOrder<Float,fineColor,1,gOrder,true,Float>;
      using gCoarse = typename gauge::FieldOrder<Float, coarseColor * coarseSpin, coarseSpin, gOrder, true, vFloat>;
      using gCoarseAtomic = typename gauge::FieldOrder<Float, coarseColor * coarseSpin, coarseSpin, gOrder, true, storeType>;

      const ColorSpinorField &v = T.Vectors(Y.Location());

      V vAccessor(const_cast<ColorSpinorField &>(v), nFace);
      F uvAccessor(*uv, nFace); // will need 2x the spin components for the KD op
      F avAccessor(*av, nFace);
      gFine gAccessor(const_cast<GaugeField &>(g));
      gFine lAccessor(const_cast<GaugeField &>(l));
      gFine xinvAccessor(const_cast<GaugeField&>(XinvKD));
      gCoarse yAccessor(const_cast<GaugeField &>(Y));
      gCoarse xAccessor(const_cast<GaugeField &>(X));
      gCoarseAtomic yAccessorAtomic(*Yatomic);
      gCoarseAtomic xAccessorAtomic(*Xatomic);

      // repeated xAccessor are just dummy values
      calculateY<QUDA_CUDA_FIELD_LOCATION, false, Float, fineSpin, fineColor, coarseSpin, coarseColor>(
        yAccessor, xAccessor, yAccessorAtomic, xAccessorAtomic, uvAccessor, avAccessor, vAccessor, gAccessor, lAccessor,
        xinvAccessor, xAccessor, xAccessor, Y, X, *Yatomic, *Xatomic, *uv, *av, v, g, l, XinvKD, *dummyClover(),
        kappa, mass, mu_dummy, mu_factor_dummy, dirac, matpc, need_bidirectional, T.fineToCoarse(Y.Location()),
        T.coarseToFine(Y.Location()));
    }

    // Clean up
    if (Yatomic != &Y) delete Yatomic;
    if (Xatomic != &X) delete Xatomic;

    if (av != nullptr && &T.Vectors(location) != av) delete av;
    if (uv != nullptr) delete uv;

  }

  // template on UV spin, which can be 1 for the non-KD ops but needs to be 2 for the KD op
  template <typename Float, typename vFloat, int fineColor, int fineSpin, int coarseColor, int coarseSpin>
  void aggregateStaggeredY(GaugeField &Y, GaugeField &X, const Transfer &T, const GaugeField &g, const GaugeField &l,
                           const GaugeField &XinvKD, double mass, QudaDiracType dirac, QudaMatPCType matpc)
  {
    if (dirac == QUDA_STAGGERED_DIRAC || dirac == QUDA_ASQTAD_DIRAC) {
      // uvSpin == 1
      aggregateStaggeredY<Float, vFloat, fineColor, fineSpin, coarseColor, coarseSpin, 1>(Y, X, T, g, l, XinvKD, mass, dirac, matpc);
    } else if (dirac == QUDA_STAGGEREDKD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC) {
      // uvSpin == 2
      aggregateStaggeredY<Float, vFloat, fineColor, fineSpin, coarseColor, coarseSpin, 2>(Y, X, T, g, l, XinvKD, mass, dirac, matpc);
    } else {
      errorQuda("Unexpected dirac type %d\n", dirac);
    }
  }

  // template on the number of coarse degrees of freedom, branch between naive K-D 
  // and actual aggregation
  template <typename Float, typename vFloat, int fineColor, int fineSpin>
  void calculateStaggeredY(GaugeField &Y, GaugeField &X, const Transfer &T, const GaugeField &g, const GaugeField &l,
                           const GaugeField &XinvKD, double mass, QudaDiracType dirac, QudaMatPCType matpc)
  {
    const int coarseSpin = 2;
    const int coarseColor = Y.Ncolor() / coarseSpin;

    if (coarseColor == 24) {
      if (T.getTransferType() == QUDA_TRANSFER_COARSE_KD)
        // the permutation routines don't need Yatomic, Xatomic, uv, av
        // need to check this sooner
        calculateStaggeredY<Float,vFloat,fineColor,fineSpin,24,coarseSpin>(Y, X, T, g, mass, dirac, matpc);
      else {
        // free field aggregation
        aggregateStaggeredY<Float,vFloat,fineColor,fineSpin,24,coarseSpin>(Y, X, T, g, l, XinvKD, mass, dirac, matpc);
      }
    } else if (coarseColor == 64) {
      aggregateStaggeredY<Float,vFloat,fineColor,fineSpin,64,coarseSpin>(Y, X, T, g, l, XinvKD, mass, dirac, matpc);
    } else if (coarseColor == 96) {
      aggregateStaggeredY<Float,vFloat,fineColor,fineSpin,96,coarseSpin>(Y, X, T, g, l, XinvKD, mass, dirac, matpc);
    } else { // revisit 3 -> 96 later
      errorQuda("Unsupported number of coarse dof %d\n", Y.Ncolor());
    }
  }

  // template on fine spin
  template <typename Float, typename vFloat, int fineColor>
  void calculateStaggeredY(GaugeField &Y, GaugeField &X, const Transfer &T, const GaugeField &g, const GaugeField &l,
                           const GaugeField &XinvKD, double mass, QudaDiracType dirac, QudaMatPCType matpc)
  {
    if (T.Vectors().Nspin() == 1) {
      calculateStaggeredY<Float,vFloat,fineColor,1>(Y, X, T, g, l, XinvKD, mass, dirac, matpc);
    } else {
      errorQuda("Unsupported number of spins %d\n", T.Vectors(X.Location()).Nspin());
    }
  }

  // template on fine colors
  template <typename Float, typename vFloat>
  void calculateStaggeredY(GaugeField &Y, GaugeField &X, const Transfer &T, const GaugeField &g, const GaugeField &l,
                           const GaugeField &XinvKD, double mass, QudaDiracType dirac, QudaMatPCType matpc)
  {
    if (g.Ncolor() == 3) {
      calculateStaggeredY<Float,vFloat,3>(Y, X, T, g, l, XinvKD, mass, dirac, matpc);
    } else {
      errorQuda("Unsupported number of colors %d\n", g.Ncolor());
    }
  }

  // template on precision of gauge links, whether or not we're coarsening the KD op, precision of KD field
  void calculateStaggeredY(GaugeField &Y, GaugeField &X, const Transfer &T, const GaugeField &g, const GaugeField &l,
                           const GaugeField &XinvKD, double mass, QudaDiracType dirac, QudaMatPCType matpc)
  {
#if defined(GPU_MULTIGRID) && defined(GPU_STAGGERED_DIRAC)
    checkPrecision(T.Vectors(X.Location()), X, Y);

    if (getVerbosity() >= QUDA_SUMMARIZE) printfQuda("Computing Y field......\n");

    if (Y.Precision() == QUDA_DOUBLE_PRECISION) {
#ifdef GPU_MULTIGRID_DOUBLE
      if (T.Vectors(X.Location()).Precision() == QUDA_DOUBLE_PRECISION) {
        calculateStaggeredY<double,double>(Y, X, T, g, l, XinvKD, mass, dirac, matpc);
      } else {
        errorQuda("Unsupported precision %d\n", Y.Precision());
      }
#else
      errorQuda("Double precision multigrid has not been enabled");
#endif
    } else 
#if QUDA_PRECISION & 4
    if (Y.Precision() == QUDA_SINGLE_PRECISION) {
      if (T.Vectors(X.Location()).Precision() == QUDA_SINGLE_PRECISION) {
        calculateStaggeredY<float,float>(Y, X, T, g, l, XinvKD, mass, dirac, matpc);
      } else {
        errorQuda("Unsupported precision %d\n", T.Vectors(X.Location()).Precision());
      }
    } else 
#endif
#if QUDA_PRECISION & 2
    if (Y.Precision() == QUDA_HALF_PRECISION) {
      if (T.Vectors(X.Location()).Precision() == QUDA_HALF_PRECISION) {
        calculateStaggeredY<float,short>(Y, X, T, g, l, XinvKD, mass, dirac, matpc);
      } else {
        errorQuda("Unsupported precision %d\n", T.Vectors(X.Location()).Precision());
      }
    } else
#endif
    {
      errorQuda("Unsupported precision %d\n", Y.Precision());
    }
    if (getVerbosity() >= QUDA_SUMMARIZE) printfQuda("....done computing Y field\n");
#else
    errorQuda("Staggered multigrid has not been built");
#endif
  }

  //Calculates the coarse color matrix and puts the result in Y.
  //N.B. Assumes Y, X have been allocated.
  void StaggeredCoarseOp(GaugeField &Y, GaugeField &X, const Transfer &T, const cudaGaugeField &gauge,
                         const cudaGaugeField &longGauge, const GaugeField &XinvKD, double mass, QudaDiracType dirac, QudaMatPCType matpc)
  {
    QudaPrecision precision = Y.Precision();
    QudaFieldLocation location = checkLocation(Y, X);

    // sanity check long link coarsening
    if ((dirac == QUDA_ASQTAD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC) && &gauge == &longGauge)
      errorQuda("Dirac type is %d but fat and long gauge links alias", dirac);

    // sanity check KD op coarsening
    if ((dirac == QUDA_STAGGEREDKD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC) && &gauge == &XinvKD)
      errorQuda("Dirac type is %d but fat links and KD inverse fields alias", dirac);

    if (dirac == QUDA_ASQTADKD_DIRAC && &longGauge == &XinvKD)
      errorQuda("Dirac type is %d but long links and KD inverse fields alias", dirac);

    if ((dirac == QUDA_STAGGEREDKD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC) && XinvKD.Reconstruct() != QUDA_RECONSTRUCT_NO)
      errorQuda("Invalid reconstruct %d for KD inverse field", XinvKD.Reconstruct());

    GaugeField *U = location == QUDA_CUDA_FIELD_LOCATION ? const_cast<cudaGaugeField*>(&gauge) : nullptr;
    GaugeField *L = (location == QUDA_CUDA_FIELD_LOCATION && (dirac == QUDA_ASQTAD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC)) ? const_cast<cudaGaugeField*>(&longGauge) : nullptr;
    GaugeField *Xinv = (location == QUDA_CUDA_FIELD_LOCATION && (dirac == QUDA_STAGGEREDKD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC)) ? const_cast<GaugeField*>(&XinvKD) : nullptr;

    if (location == QUDA_CPU_FIELD_LOCATION) {
      //First make a cpu gauge field from the cuda gauge field
      int pad = 0;
      GaugeFieldParam gf_param(gauge.X(), precision, QUDA_RECONSTRUCT_NO, pad, gauge.Geometry());
      gf_param.order = QUDA_QDP_GAUGE_ORDER;
      gf_param.fixed = gauge.GaugeFixed();
      gf_param.link_type = gauge.LinkType();
      gf_param.t_boundary = gauge.TBoundary();
      gf_param.anisotropy = gauge.Anisotropy();
      gf_param.gauge = nullptr;
      gf_param.create = QUDA_NULL_FIELD_CREATE;
      gf_param.siteSubset = QUDA_FULL_SITE_SUBSET;
      gf_param.nFace = 1;
      gf_param.ghostExchange = QUDA_GHOST_EXCHANGE_PAD;

      U = new cpuGaugeField(gf_param);

      //Copy the cuda gauge field to the cpu
      gauge.saveCPUField(*static_cast<cpuGaugeField*>(U));

      // Create either a real or a dummy L field
      GaugeFieldParam lgf_param(longGauge.X(), precision, QUDA_RECONSTRUCT_NO, pad, longGauge.Geometry());
      if (!(dirac == QUDA_ASQTAD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC))
        for (int i = 0; i < lgf_param.nDim; i++) lgf_param.x[i] = 0;
      lgf_param.order = QUDA_QDP_GAUGE_ORDER;
      lgf_param.fixed = longGauge.GaugeFixed();
      lgf_param.link_type = longGauge.LinkType();
      lgf_param.t_boundary = longGauge.TBoundary();
      lgf_param.anisotropy = longGauge.Anisotropy();
      lgf_param.gauge = nullptr;
      lgf_param.create = QUDA_NULL_FIELD_CREATE;
      lgf_param.siteSubset = QUDA_FULL_SITE_SUBSET;
      lgf_param.nFace = 3;
      lgf_param.ghostExchange = QUDA_GHOST_EXCHANGE_PAD;

      L = new cpuGaugeField(lgf_param);

      //Copy the cuda gauge field to the cpu
      if (dirac == QUDA_ASQTAD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC)
        longGauge.saveCPUField(*static_cast<cpuGaugeField*>(L));
      
      // Create either a real or a dummy Xinv field
      GaugeFieldParam xgf_param(XinvKD.X(), precision, QUDA_RECONSTRUCT_NO, pad, XinvKD.Geometry());
      if (!(dirac == QUDA_STAGGEREDKD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC))
        for (int i = 0; i < xgf_param.nDim; i++) xgf_param.x[i] = 0;
      xgf_param.order = QUDA_QDP_GAUGE_ORDER;
      xgf_param.fixed = XinvKD.GaugeFixed();
      xgf_param.link_type = XinvKD.LinkType();
      xgf_param.t_boundary = XinvKD.TBoundary();
      xgf_param.anisotropy = XinvKD.Anisotropy();
      if (dirac == QUDA_STAGGEREDKD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC) {
        xgf_param.create = QUDA_COPY_FIELD_CREATE;
      } else {
        xgf_param.gauge = nullptr;
        xgf_param.create = QUDA_NULL_FIELD_CREATE;
      }
      xgf_param.siteSubset = QUDA_FULL_SITE_SUBSET;
      xgf_param.nFace = 0;
      xgf_param.ghostExchange = QUDA_GHOST_EXCHANGE_NO;

      Xinv = new cpuGaugeField(xgf_param);

      //Copy the cuda gauge field to the cpu
      //if (dirac == QUDA_STAGGEREDKD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC)
      //  XinvKD.saveCPUField(*static_cast<cpuGaugeField*>(Xinv));

      
    } else if (location == QUDA_CUDA_FIELD_LOCATION) {

      int pad = 0;

      // create some dummy fields if need be
      if (!(dirac == QUDA_ASQTAD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC)) {
        // create a dummy field
        GaugeFieldParam lgf_param(longGauge);
        for (int i = 0; i < lgf_param.nDim; i++) lgf_param.x[i] = 0;
        lgf_param.reconstruct = QUDA_RECONSTRUCT_NO;
        lgf_param.order = QUDA_FLOAT2_GAUGE_ORDER;
        lgf_param.setPrecision(lgf_param.Precision());
        lgf_param.create = QUDA_NULL_FIELD_CREATE;
        L = new cudaGaugeField(lgf_param);
      } else if ((dirac == QUDA_ASQTAD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC) && longGauge.Reconstruct() != QUDA_RECONSTRUCT_NO) {
        // create a copy of the gauge field with no reconstruction
        GaugeFieldParam lgf_param(longGauge);
        lgf_param.reconstruct = QUDA_RECONSTRUCT_NO;
        lgf_param.order = QUDA_FLOAT2_GAUGE_ORDER;
        lgf_param.setPrecision(lgf_param.Precision());
        L = new cudaGaugeField(lgf_param);

        L->copy(longGauge);
      }

      if (!(dirac == QUDA_STAGGEREDKD_DIRAC || dirac == QUDA_ASQTADKD_DIRAC)) {
        // Create a dummy field
        GaugeFieldParam xgf_param(XinvKD.X(), precision, QUDA_RECONSTRUCT_NO, pad, XinvKD.Geometry());
        for (int i = 0; i < xgf_param.nDim; i++) xgf_param.x[i] = 0;
        xgf_param.reconstruct = QUDA_RECONSTRUCT_NO;
        xgf_param.order = QUDA_FLOAT2_GAUGE_ORDER;
        xgf_param.setPrecision(xgf_param.Precision());
        xgf_param.create = QUDA_NULL_FIELD_CREATE;
        Xinv = new cudaGaugeField(xgf_param);
      }
      // no need to worry about XinvKD's reconstruct

      if (gauge.Reconstruct() != QUDA_RECONSTRUCT_NO) {
        //Create a copy of the gauge field with no reconstruction, required for fine-grained access
        GaugeFieldParam gf_param(gauge);
        gf_param.reconstruct = QUDA_RECONSTRUCT_NO;
        gf_param.order = QUDA_FLOAT2_GAUGE_ORDER;
        gf_param.setPrecision(gf_param.Precision());
        U = new cudaGaugeField(gf_param);

        U->copy(gauge);
      }
      
    }

    calculateStaggeredY(Y, X, T, *U, *L, *Xinv, mass, dirac, matpc);

    if (U != &gauge) delete U;
    if (L != &longGauge) delete L;
    if (Xinv != &XinvKD) delete Xinv;
  }

} //namespace quda
